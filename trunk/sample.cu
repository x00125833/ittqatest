
#include <hip/hip_runtime.h>
struct cuBCData {
    int* distances;      // size n
    int* numSPs;         // size n
    bool* predecessor;   // size 2m
    float* dependencies; // size n
    float* nodeBCs;      // size n
};

/*
 * global data structure across all blocks
 */
struct cuGraph {
    int* froms; // size 2m
    int* nhbrs; // size 2m
};

__global__ void edge_bc_bfs(int nedges, cuGraph* cuGlobalGraph, cuBCData* bcd) {
  
  __shared__ bool done;
  
  if(threadIdx.x == 0){
    done = true;
  }

  int d = 0;

  while (!done) {
    __syncthreads();
    done = true;        //not sure what this does
    d++;                //are all threads executing these statements?
    __syncthreads();

    for (int eid = threadIdx.x; eid < 2 * nedges; eid += blockDim.x) {
      int from = cuGlobalGraph->froms[eid];
      if (bcd->distances[from] == d) {
        int nhbr = cuGlobalGraph->nhbrs[eid];
        int nhbr_dist = bcd->distances[nhbr];
        if (nhbr_dist == -1) {
          bcd->distances[nhbr] = nhbr_dist = d + 1;
          done = false;
        } else if (nhbr_dist < d) {
          bcd->predecessor[eid] = true;
        }
        if (nhbr_dist == d + 1) {
          atomicAdd(&bcd->numSPs[nhbr], bcd->numSPs[from]);
        }
      }
      __syncthreads();
    }
  }
}
  